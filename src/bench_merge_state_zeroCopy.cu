/*
 * Copyright (c) 2023 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
 #include <thrust/device_vector.h>

 #include <cstddef>
 #include <flashinfer/attention/cascade.cuh>
 #include <nvbench/nvbench.cuh>
 
 #include "flashinfer_ops.cuh"
 #include "utils.h"
 
 using namespace flashinfer;
 
  
#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
    }                                                                          \
}
   
 
 
 /*!
  * \brief Merge the self-attention state with another state in place.
  * \tparam DType The data type of v and v_other.
  * \param v The partial v to be updated in-place. (n, h, d)
  * \param s The logsumexp value to be updated in-place. (n, h)
  * \param v_other The other v to be merged. (n, h, d)
  * \param s_other The other logsumexp value to be merged. (n, h)
  * \param seq_len The sequence length.
  * \param num_heads The number of heads of v and v_other.
  * \param head_dim The dimension of each head.
  * \param mask Optional mask of whether to merge given sequences or not. (n)
  * \param stream The CUDA stream to execute the kernel.
  * \return status Indicates whether CUDA calls are successful
  * \note Both s and s_other are logsumexp values with base 2.
  */
 
 template <typename T>
 void bench_merge_state_zeroCopy(nvbench::state& state) {
    
   const auto seq_len = state.get_int64("seq_len");
   const auto num_heads = state.get_int64("num_heads");
   const auto head_dim = state.get_int64("head_dim");
 
   std::vector<T> V_a_host_(seq_len  * num_heads * head_dim);
   std::vector<float> S_a_host_(seq_len  * num_heads);
 
   std::vector<T> V_b_host_(seq_len  * num_heads * head_dim);
   std::vector<float> S_b_host_(seq_len  * num_heads);
 
   utils::vec_normal_(V_a_host_);
   utils::vec_uniform_(S_a_host_, 5, 10);
   utils::vec_normal_(V_b_host_);
   utils::vec_uniform_(S_b_host_, 5, 10);
    
   thrust::host_vector<T> V_a_host(V_a_host_);
   thrust::host_vector<float> S_a_host(S_a_host_);
   thrust::host_vector<T> V_b_host(V_b_host_);
   thrust::host_vector<float> S_b_host(S_b_host_);
   CHECK(hipHostRegister(V_a_host.data(), sizeof(T) * seq_len  * num_heads * head_dim, hipHostRegisterDefault));
   CHECK(hipHostRegister(S_a_host.data(), sizeof(float) * seq_len  * num_heads, hipHostRegisterDefault));
   CHECK(hipHostRegister(V_b_host.data(), sizeof(T) * seq_len  * num_heads * head_dim, hipHostRegisterDefault));
   CHECK(hipHostRegister(S_b_host.data(), sizeof(float) * seq_len  * num_heads, hipHostRegisterDefault));
   
   thrust::device_vector<T> V_merged(seq_len * num_heads * head_dim);
   thrust::device_vector<float> S_merged(seq_len * num_heads);
 
   //  state.add_global_memory_reads<T>(V_a_host.size(), "Read");
   state.add_global_memory_reads<T>(V_merged.size(), "Write");
   state.add_global_memory_writes<T>(S_merged.size(), "Write");
 
   state.exec(nvbench::exec_tag::timer, [&](nvbench::launch& launch, auto& timer) {
     timer.start();
     hipError_t status = MergeState(
        V_a_host.data(), S_a_host.data(),
        V_b_host.data(), S_b_host.data(),
        thrust::raw_pointer_cast(V_merged.data()), thrust::raw_pointer_cast(S_merged.data()),
        seq_len, num_heads, head_dim);
     timer.stop();
   });
   CHECK(hipDeviceSynchronize());
   CHECK(hipHostUnregister(V_a_host.data() ));
   CHECK(hipHostUnregister(S_a_host.data() ));
   CHECK(hipHostUnregister(V_b_host.data() ));
   CHECK(hipHostUnregister(S_b_host.data() ));
 }
   
 
 
 
 /*!
  * \brief Merge the self-attention state with another state in place.
  * \tparam DType The data type of v and v_other.
  * \param v The partial v to be updated in-place. (n, h, d)
  * \param s The logsumexp value to be updated in-place. (n, h)
  * \param v_other The other v to be merged. (n, h, d)
  * \param s_other The other logsumexp value to be merged. (n, h)
  * \param seq_len The sequence length.
  * \param num_heads The number of heads of v and v_other.
  * \param head_dim The dimension of each head.
  * \param mask Optional mask of whether to merge given sequences or not. (n)
  * \param stream The CUDA stream to execute the kernel.
  * \return status Indicates whether CUDA calls are successful
  * \note Both s and s_other are logsumexp values with base 2.
  */
 template <typename T>
 void bench_merge_state_InPlace_zeroCopy(nvbench::state& state) {
    
    const auto seq_len = state.get_int64("seq_len");
    const auto num_heads = state.get_int64("num_heads");
    const auto head_dim = state.get_int64("head_dim");

    std::vector<T> V_a_host_(seq_len  * num_heads * head_dim);
    std::vector<float> S_a_host_(seq_len  * num_heads);

    std::vector<T> V_other_host(seq_len  * num_heads * head_dim);
    std::vector<float> S_other_host(seq_len  * num_heads);

    utils::vec_normal_(V_a_host_);
    utils::vec_uniform_(S_a_host_, 5, 10);
    utils::vec_normal_(V_other_host);
    utils::vec_uniform_(S_other_host, 5, 10);
      
    thrust::host_vector<T> V_a_host(V_a_host_);
    thrust::host_vector<float> S_a_host(S_a_host_);
    CHECK(hipHostRegister(V_a_host.data(), sizeof(T) * seq_len  * num_heads * head_dim, hipHostRegisterDefault));
    CHECK(hipHostRegister(S_a_host.data(), sizeof(float) * seq_len  * num_heads, hipHostRegisterDefault));
    thrust::device_vector<T> V_other_device(V_other_host);
    thrust::device_vector<float> S_other_device(S_other_host);


    //  state.add_global_memory_reads<T>(V_a_host.size(), "Read");
    state.add_global_memory_writes<T>(V_other_device.size(), "Write");
    state.add_global_memory_writes<T>(S_other_device.size(), "Write");

    // state.exec(nvbench::exec_tag::sync, <KernelLauncher>); // Safe
    state.exec(nvbench::exec_tag::timer, [&](nvbench::launch& launch, auto& timer) {
      timer.start();
      hipError_t status = MergeStateInPlace(
          V_a_host.data(), S_a_host.data(),
          thrust::raw_pointer_cast(V_other_device.data()), thrust::raw_pointer_cast(S_other_device.data()),
          seq_len, num_heads, head_dim);
      timer.stop();
   });
   CHECK(hipDeviceSynchronize());
   CHECK(hipHostUnregister(V_a_host.data() ));
   CHECK(hipHostUnregister(S_a_host.data() ));
 }
   


 #define STR_HELPER(x) #x
 #define STR(x) STR_HELPER(x)
  
 
 #define BENCH_FLASHINFER_MERGE_STATE_KERNELS(T)                            \
 auto bench_flashinfer_merge_state_zeroCopy_##T##_ = bench_merge_state_zeroCopy<T>; \
 NVBENCH_BENCH(bench_flashinfer_merge_state_zeroCopy_##T##_)                \
     .set_name("flashinfer_merge_state_zeroCopy_" STR(T))                   \
     .add_int64_axis("seq_len", {1, 2, 4, 8, 16, 32, 64, 128, 256}) \
     .add_int64_axis("num_heads", {32,56})                             \
     .add_int64_axis("head_dim", {64,128})
 
 
 
 
 #define BENCH_FLASHINFER_MERGE_STATE_InPlace_KERNELS(T)                            \
 auto bench_flashinfer_merge_state_InPlace_zeroCopy_##T##_ = bench_merge_state_InPlace_zeroCopy<T>; \
 NVBENCH_BENCH(bench_flashinfer_merge_state_InPlace_zeroCopy_##T##_)                \
     .set_name("flashinfer_merge_state_InPlace_zeroCopy_" STR(T))                   \
     .add_int64_axis("seq_len", {1, 2, 4, 8, 16, 32, 64, 128, 256}) \
     .add_int64_axis("num_heads", {32,56})                             \
     .add_int64_axis("head_dim", {64,128})


BENCH_FLASHINFER_MERGE_STATE_KERNELS(half); 
BENCH_FLASHINFER_MERGE_STATE_InPlace_KERNELS(half); 