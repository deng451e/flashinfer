/*
 * Copyright (c) 2023 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <thrust/device_vector.h>

#include <cstddef>
#include <flashinfer/attention/cascade.cuh>
#include <nvbench/nvbench.cuh>

#include "flashinfer_ops.cuh"
#include "utils.h"

using namespace flashinfer;

 

template <typename T>
void bench_merge_states(nvbench::state& state) {
  const auto num_index_sets = state.get_int64("num_index_sets");
  const auto seq_len = state.get_int64("seq_len");
  const auto num_heads = state.get_int64("num_heads");
  const auto head_dim = state.get_int64("head_dim");

  std::vector<T> V_a_host(seq_len * num_index_sets * num_heads * head_dim);
  std::vector<float> S_a_host(seq_len * num_index_sets * num_heads);

  utils::vec_normal_(V_a_host);
  utils::vec_uniform_(S_a_host, 5, 10);

  thrust::device_vector<T> V_a_device(V_a_host);
  thrust::device_vector<float> S_a_device(S_a_host);
  thrust::device_vector<T> V_merged(seq_len * num_heads * head_dim);
  thrust::device_vector<float> S_merged(seq_len * num_heads);

  state.add_global_memory_reads<T>(V_a_host.size(), "Read");
  state.add_global_memory_writes<T>(V_merged.size(), "Write");

  state.exec(nvbench::exec_tag::timer, [&](nvbench::launch& launch, auto& timer) {
    timer.start();
    hipError_t status = MergeStates(
        thrust::raw_pointer_cast(V_a_device.data()), thrust::raw_pointer_cast(S_a_device.data()),
        thrust::raw_pointer_cast(V_merged.data()), thrust::raw_pointer_cast(S_merged.data()),
        num_index_sets, seq_len, num_heads, head_dim);
    timer.stop();
  });
}



/*!
 * \brief Merge the self-attention state with another state in place.
 * \tparam DType The data type of v and v_other.
 * \param v The partial v to be updated in-place. (n, h, d)
 * \param s The logsumexp value to be updated in-place. (n, h)
 * \param v_other The other v to be merged. (n, h, d)
 * \param s_other The other logsumexp value to be merged. (n, h)
 * \param seq_len The sequence length.
 * \param num_heads The number of heads of v and v_other.
 * \param head_dim The dimension of each head.
 * \param mask Optional mask of whether to merge given sequences or not. (n)
 * \param stream The CUDA stream to execute the kernel.
 * \return status Indicates whether CUDA calls are successful
 * \note Both s and s_other are logsumexp values with base 2.
 */

template <typename T>
void bench_merge_state(nvbench::state& state) {
   
  const auto seq_len = state.get_int64("seq_len");
  const auto num_heads = state.get_int64("num_heads");
  const auto head_dim = state.get_int64("head_dim");

  std::vector<T> V_a_host(seq_len  * num_heads * head_dim);
  std::vector<float> S_a_host(seq_len  * num_heads);

  std::vector<T> V_b_host(seq_len  * num_heads * head_dim);
  std::vector<float> S_b_host(seq_len  * num_heads);

  utils::vec_normal_(V_a_host);
  utils::vec_uniform_(S_a_host, 5, 10);
  utils::vec_normal_(V_b_host);
  utils::vec_uniform_(S_b_host, 5, 10);

  thrust::device_vector<T> V_a_device(V_a_host);
  thrust::device_vector<float> S_a_device(S_a_host);
  thrust::device_vector<T> V_b_device(V_b_host);
  thrust::device_vector<float> S_b_device(S_b_host);
  thrust::device_vector<T> V_merged(seq_len * num_heads * head_dim);
  thrust::device_vector<float> S_merged(seq_len * num_heads);

  state.add_global_memory_reads<T>(V_a_host.size(), "Read");
  state.add_global_memory_writes<T>(V_merged.size(), "Write");

  state.exec(nvbench::exec_tag::timer, [&](nvbench::launch& launch, auto& timer) {
    timer.start();
    hipError_t status = MergeState(
        thrust::raw_pointer_cast(V_a_device.data()), thrust::raw_pointer_cast(S_a_device.data()),
        thrust::raw_pointer_cast(V_b_device.data()), thrust::raw_pointer_cast(S_b_device.data()),
        thrust::raw_pointer_cast(V_merged.data()), thrust::raw_pointer_cast(S_merged.data()),
        seq_len, num_heads, head_dim);
    timer.stop();
  });
}
  



/*!
 * \brief Merge the self-attention state with another state in place.
 * \tparam DType The data type of v and v_other.
 * \param v The partial v to be updated in-place. (n, h, d)
 * \param s The logsumexp value to be updated in-place. (n, h)
 * \param v_other The other v to be merged. (n, h, d)
 * \param s_other The other logsumexp value to be merged. (n, h)
 * \param seq_len The sequence length.
 * \param num_heads The number of heads of v and v_other.
 * \param head_dim The dimension of each head.
 * \param mask Optional mask of whether to merge given sequences or not. (n)
 * \param stream The CUDA stream to execute the kernel.
 * \return status Indicates whether CUDA calls are successful
 * \note Both s and s_other are logsumexp values with base 2.
 */
template <typename T>
void bench_merge_state_InPlace(nvbench::state& state) {
   
  const auto seq_len = state.get_int64("seq_len");
  const auto num_heads = state.get_int64("num_heads");
  const auto head_dim = state.get_int64("head_dim");

  std::vector<T> V_a_host(seq_len  * num_heads * head_dim);
  std::vector<float> S_a_host(seq_len  * num_heads);

  std::vector<T> V_other_host(seq_len  * num_heads * head_dim);
  std::vector<float> S_other_host(seq_len  * num_heads);

  utils::vec_normal_(V_a_host);
  utils::vec_uniform_(S_a_host, 5, 10);
  utils::vec_normal_(V_other_host);
  utils::vec_uniform_(S_other_host, 5, 10);

  thrust::device_vector<T> V_a_device(V_a_host);
  thrust::device_vector<float> S_a_device(S_a_host);
  thrust::device_vector<T> V_other_device(V_other_host);
  thrust::device_vector<float> S_other_device(S_other_host);
 

  state.add_global_memory_reads<T>(V_a_host.size(), "Read");
  state.add_global_memory_writes<T>(V_other_device.size(), "Write");

  state.exec(nvbench::exec_tag::timer, [&](nvbench::launch& launch, auto& timer) {
    timer.start();
    hipError_t status = MergeStateInPlace(
        thrust::raw_pointer_cast(V_a_device.data()), thrust::raw_pointer_cast(S_a_device.data()),
        thrust::raw_pointer_cast(V_other_device.data()), thrust::raw_pointer_cast(S_other_device.data()),
        seq_len, num_heads, head_dim);
    timer.stop();
  });
}
  
#define STR_HELPER(x) #x
#define STR(x) STR_HELPER(x)


#define BENCH_FLASHINFER_MERGE_STATES_KERNELS(T)                            \
  auto bench_flashinfer_merge_states_##T##_ = bench_merge_states<T>; \
  NVBENCH_BENCH(bench_flashinfer_merge_states_##T##_)                \
      .set_name("flashinfer_merge_states_" STR(T))                   \
      .add_int64_axis("num_index_sets", {2, 16, 64, 128, 256})       \
      .add_int64_axis("seq_len", {1, 2, 4, 8, 16, 32, 64, 128, 256}) \
      .add_int64_axis("num_heads", {32})                             \
      .add_int64_axis("head_dim", {128})



#define BENCH_FLASHINFER_MERGE_STATE_KERNELS(T)                            \
auto bench_flashinfer_merge_state_##T##_ = bench_merge_state<T>; \
NVBENCH_BENCH(bench_flashinfer_merge_state_##T##_)                \
    .set_name("flashinfer_merge_state_" STR(T))                   \
    .add_int64_axis("seq_len", {1, 2, 4, 8, 16, 32, 64, 128, 256}) \
    .add_int64_axis("num_heads", {32,56})                             \
    .add_int64_axis("head_dim", {64,80,128})




#define BENCH_FLASHINFER_MERGE_STATE_InPlace_KERNELS(T)                            \
auto bench_flashinfer_merge_state_InPlace_##T##_ = bench_merge_state_InPlace<T>; \
NVBENCH_BENCH(bench_flashinfer_merge_state_InPlace_##T##_)                \
    .set_name("flashinfer_merge_state_InPlace_" STR(T))                   \
    .add_int64_axis("seq_len", {1, 2, 4, 8, 16, 32, 64, 128, 256}) \
    .add_int64_axis("num_heads", {32,56})                             \
    .add_int64_axis("head_dim", {64,80,128})


  
// BENCH_FLASHINFER_MERGE_STATES_KERNELS(half); 
BENCH_FLASHINFER_MERGE_STATE_KERNELS(half); 
BENCH_FLASHINFER_MERGE_STATE_InPlace_KERNELS(half); 